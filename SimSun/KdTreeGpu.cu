#include "hip/hip_runtime.h"
#include "Configuration.h"

#include "BoundingBox.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "Vector3.h"
#include "Ray.h"
#include "BenchmarkTimer.h"
#include "KdTreeGpu.h"
#include "Result.h"

extern BenchmarkTimer gpuTotalTime;
extern BenchmarkTimer gpuTransferTime;

void CUDA_VALIDATE(hipError_t e)
{
	if(e != hipSuccess)
	{
		fprintf(stderr, "Cuda Error: %d\n Exiting.", e);
		exit(e);
	}
}

__device__ double getField(const Vector3 & vector, Axis a)
{
	switch(a)
	{
	case X:
		return vector.x;
	case Y:
		return vector.y;
	}
	return vector.z;
}

/*
Returns the position of the ray o+td at point t, moved just sligtly in the direction of reflectedDirection 
to avoid to hit the same location again
*/

__device__ Vector3 rayPosition(const Vector3 & rayOrigin,  const Vector3 & rayDirection, const double t, const Vector3 & reflectedDirection)
{
	Vector3 a = rayOrigin;
	a.x += t*rayDirection.x + 0.00001*reflectedDirection.x;
	a.y += t*rayDirection.y + 0.00001*reflectedDirection.y;
	a.z += t*rayDirection.z + 0.00001*reflectedDirection.z;
	return a;
}

/*
 Reflect function
 Reflect this direction along normal, returns reflected vector
*/

__device__ double vectorDot(const Vector3& a, const Vector3& b)
{
	return a.x*b.x + a.y*b.y + a.z*b.z;
}

__device__ Vector3 reflectVector(const Vector3& direction, const Vector3 & normal)
{
	Vector3 result = direction;
	double dotP = vectorDot(direction, normal);
	result.x = direction.x - 2 * normal.x * dotP;
	result.y = direction.y - 2 * normal.y * dotP;
	result.z = direction.z - 2 * normal.z * dotP;
	return result;
}

/*
Find triangle intersections
*/

__device__ Vector3 rtIntersectionfindPoint(const Vector3 & a, const Vector3 & b, const double bScale, const Vector3 & c)
{
	Vector3 out = a;
	out.x = a.x + b.x*bScale - c.x;
	out.y = a.y + b.y*bScale - c.y;
	out.z = a.z + b.z*bScale - c.z;
	return out;
}

__device__ bool rayTriangleIntersection(const Vector3 & rayOrigin, const Vector3 & rayDirection, double tNear, double tFar,
										const TrianglePanelPair & triangle, double & tOut, Vector3 & reflection)
{

	Triangle tri = triangle.tri;

	double d = 1.0/(getField(rayDirection, tri.nAxis) + tri.nu * getField(rayDirection, tri.uAxis) + tri.nv * getField(rayDirection, tri.vAxis));
	double t = (tri.nd - (getField(rayOrigin, tri.nAxis) + tri.nu * getField(rayOrigin, tri.uAxis) + tri.nv * getField(rayOrigin, tri.vAxis))) * d;
	
	if(t >= tNear && t <= tFar)
	{
		Vector3 P = rtIntersectionfindPoint(rayOrigin, rayDirection, t, tri.p1);

		double Pu = getField(P, tri.uAxis);
		double Pv = getField(P, tri.vAxis);

		double beta = Pv * tri.bun + Pu * tri.bvn;
		if(beta < 0)
		{
			return false;
		}

		double gamma = Pu * tri.cun + Pv * tri.cvn;
		if(gamma < 0 || gamma + beta > 1)
		{
			return false;
		}

		tOut = t;
		reflection = reflectVector(rayDirection, tri.normal_normalized);
		return true;
	}

	
	return false;
}

__device__ bool rayTrianglesIntersection(const Vector3 & rayOrigin, const Vector3 & rayDirection, double & tNear, double & tFar,
										 TrianglePanelPair* triangles, int trianglesListFrom, int trianglesListTo,
										double & t, int & hitIndex, Vector3 & reflection)
{
	int triangleIndexCandidate = -1;
	double tMinCandidate = 1E999;
	Vector3 reflectionCandidate;

	for(int i = trianglesListFrom; i < trianglesListTo; i++)
	{
		double tempT;
		Vector3 tempReflection;

		if(rayTriangleIntersection(rayOrigin, rayDirection, tNear, tFar, triangles[i], tempT, tempReflection))
		{
			if(tempT < tMinCandidate)
			{
				triangleIndexCandidate = i;
				tMinCandidate = tempT;
				reflectionCandidate = tempReflection;
			}
		}
	}

	if(triangleIndexCandidate >= 0)
	{
		t = tMinCandidate;
		hitIndex = triangleIndexCandidate;
		reflection = reflectionCandidate;
		return true;
	}
	return false;
}


/*
Intersects a Bounding Box with a ray. If the ray intersects, then we know that tNear < tFar. If it does not
exist, then we make sure the opposite is true.
*/
__device__ double2 rayBoxIntersection( BoundingBox sceneBB, const Vector3 & rayOrigin, const Vector3 & rayDirection)
{

	double2 tNearFar;

	// X

	double divx = 1 / rayDirection.x;
	double tNear, tFar;
	if (divx >= 0)
	{
		tNear = (sceneBB.mi.x- rayOrigin.x) * divx;
		tFar = (sceneBB.ma.x - rayOrigin.x) * divx;
	}
	else
	{
		tNear = (sceneBB.ma.x - rayOrigin.x) * divx;
		tFar = (sceneBB.mi.x - rayOrigin.x) * divx;
	}

	if(tFar < tNear )
	{
		tNearFar.x = 1;
		tNearFar.y = 0;
		return tNearFar;
	}

	// Y

	double divy = 1 / rayDirection.y;
	double tyNear, tyFar;
	if (divy >= 0)
	{
		tyNear = (sceneBB.mi.y- rayOrigin.y) * divy;
		tyFar = (sceneBB.ma.y - rayOrigin.y) * divy;
	}
	else {
		tyNear = (sceneBB.ma.y - rayOrigin.y) * divy;
		tyFar = (sceneBB.mi.y - rayOrigin.y) * divy;
	}

	if(tyFar < tyNear )
	{
		tNearFar.x = 1;
		tNearFar.y = 0;
		return tNearFar;
	}

	if(tyNear > tNear)
	{
		tNear = tyNear;
	}

	if(tyFar < tFar)
	{
		tFar = tyFar;
	}

	// Z

	double divz = 1 / rayDirection.z;
	double tzNear, tzFar;
	if (divz >= 0)
	{
		tzNear = (sceneBB.mi.z- rayOrigin.z) * divz;
		tzFar = (sceneBB.ma.z - rayOrigin.z) * divz;
	}
	else {
		tzNear = (sceneBB.ma.z - rayOrigin.z) * divz;
		tzFar = (sceneBB.mi.z - rayOrigin.z) * divz;
	}

	if(tzFar < tzNear )
	{
		tNearFar.x = 1;
		tNearFar.y = 0;
		return tNearFar;
	}

	if(tzNear > tNear)
	{
		tNear = tzNear;
	}

	if(tzFar < tFar)
	{
		tFar = tzFar;
	}

	if(tNear > 1E200 || tFar < 0 )
	{
		tNearFar.x = 1;
		tNearFar.y = 0;
		return tNearFar;
	}
	
	tNearFar.x = tNear;
	tNearFar.y = tFar;
	return tNearFar;
}

/*
Entry Kernel
*/

__global__ void kdTreeTraversal(BoundingBox sceneBB, Ray* rays, KdTreeGpuNode* nodes, TrianglePanelPair* triangles, 
								 int firstRayIndex, int numRays, int rank, int rootNode, int* hitPanelOut, GpuResult* results)
{
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

	if(index < numRays)
	{
		if(rank > 0 && hitPanelOut[index] == -1)
		{
			return;
		}

		hitPanelOut[index] = -1;
		Ray myRay = rays[index];
		Vector3 rayOrigin = myRay.origin;
		Vector3 rayDirection = myRay.direction;

		double2 tNearFar = rayBoxIntersection(sceneBB, rayOrigin, rayDirection);
		double tNear = tNearFar.x;
		double tFar = tNearFar.y;
		const double globalTFar = tFar;
		double pushdownNode = rootNode;
		bool pushDown = true;

		if(tNear < tFar) // Ray intersects box
		{
			if(tNear < 0)
			{
				tNear = 0;
			}

			int currentNode = rootNode;

			while(true)
			{

				while(nodes[currentNode].left != -1)
				{

					double nodeS = nodes[currentNode].s;
					Axis nodeAxis = nodes[currentNode].axis;
					double rayOriginAxis = getField(rayOrigin, nodeAxis);
					double rayDirectionAxis = getField(rayDirection, nodeAxis);
					double tSplit = (nodeS - rayOriginAxis) / rayDirectionAxis;

					int nearNode = nodes[currentNode].left, farNode = nodes[currentNode].right;
					if(rayDirectionAxis < 0)
					{
						int temp = nearNode;
						nearNode = farNode;
						farNode = temp;
					}

					// Evaluate children nodes
					if (tSplit >= tFar) // Near
					{ 
						currentNode = nearNode;
					}
					else if (tSplit <= tNear) // Far
					{
						currentNode = farNode;
						
					}
					else // Near then Far
					{ 
						currentNode = nearNode;
						tFar = tSplit;
						pushDown = false;
					}

					if(pushDown)
					{
						pushdownNode = currentNode;
					}
				}

				// Check Triangles for Intersection

				int trianglesListFrom = nodes[currentNode].globalListFirst;
				int trianglesListSize = nodes[currentNode].globalListSize;

				double tClosest;
				int hitPanelIndex;
				Vector3 reflection;

				if(rayTrianglesIntersection(rayOrigin, rayDirection, tNear, tFar,
											triangles, trianglesListFrom, trianglesListFrom+trianglesListSize,
											tClosest, hitPanelIndex, reflection))
				{
					if(tClosest >= tNear && tClosest <= tFar ) 
					{	
						// Check that we have hit on the correct side of closest triangle
						if(vectorDot(triangles[hitPanelIndex].tri.normal_normalized, rayDirection) < 0)
						{
							// Return result hit
							Ray reflectedRay;
							reflectedRay.origin = rayPosition(rayOrigin, rayDirection, tClosest, reflection);
							reflectedRay.direction = reflection;

							hitPanelOut[index] = hitPanelIndex;

							results[index].reflectedRay = reflectedRay;
							results[index].t = tClosest;
							results[index].panel = triangles[hitPanelIndex].panel;
							rays[index] = reflectedRay;
						}

						return;
					}
				}

				// Continue Search

				if(tFar == globalTFar)
				{
					return;
				}
				else
				{
					currentNode = pushdownNode;
					tNear = tFar;
					tFar = globalTFar;
				}
			}
		}
	}
}

/*
 Start tracing of a RayList on the GPU. The result can later be fetched, so CPU work can be
 done in the mean time.
*/

void KdTreeGpu::traceRaysOnGpuAsync(int firstRayIndex, int numRays, int rank, int buffer)
{
	int per_block = 128;
	int num_blocks = numRays/per_block + (numRays%per_block==0?0:1);

	Ray* rays = &this->deviceRayPtr[firstRayIndex];
	int* deviceLocalHitPanelIds = &this->deviceHitPanelIdPtr[firstRayIndex];

	kdTreeTraversal<<<num_blocks, per_block, 0>>>(sceneBoundingBox, rays, deviceNodesPtr, deviceTrianglesListPtr, 
											 	firstRayIndex, numRays, rank, rootNodeIndex, 
												deviceLocalHitPanelIds, deviceResults);

	hipStreamQuery(0);
	CUDA_VALIDATE(hipMemcpyAsync(resultHitPanelIds[buffer], deviceLocalHitPanelIds, numRays*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_VALIDATE(hipMemcpyAsync(results[buffer], deviceResults, numRays*sizeof(GpuResult), hipMemcpyDeviceToHost));
	hipStreamQuery(0);
}

void KdTreeGpu::transferRaysToGpu(RayList & rays)
{
	gpuTransferTime.start();
	CUDA_VALIDATE(hipMalloc(&this->deviceRayPtr, rays.size()*sizeof(Ray)));
	CUDA_VALIDATE(hipMalloc(&this->deviceHitPanelIdPtr, rays.size()*sizeof(int)));
	CUDA_VALIDATE(hipMemcpyAsync(this->deviceRayPtr, &rays[0], rays.size()*sizeof(Ray), hipMemcpyHostToDevice));
	gpuTransferTime.stop();
}

void KdTreeGpu::waitForGpu()
{
	CUDA_VALIDATE(hipDeviceSynchronize());
}

/*
Allocating pinned memory on the host that can be accessed asynchronously by the GPU
*/

void KdTreeGpu::allocateHostResultBuffers()
{

	unsigned int flag = hipHostMallocPortable;

	CUDA_VALIDATE(hipHostAlloc(&resultHitPanelIds[0], MAX_RAYS_PER_ITERATION*sizeof(int), flag));
	CUDA_VALIDATE(hipHostAlloc(&resultHitPanelIds[1], MAX_RAYS_PER_ITERATION*sizeof(int), flag));

	CUDA_VALIDATE(hipHostAlloc(&results[0], MAX_RAYS_PER_ITERATION*sizeof(GpuResult), flag));
	CUDA_VALIDATE(hipHostAlloc(&results[1], MAX_RAYS_PER_ITERATION*sizeof(GpuResult), flag));
}

void KdTreeGpu::freeHostResultBuffers()
{
	CUDA_VALIDATE(hipHostFree(resultHitPanelIds[0]));
	CUDA_VALIDATE(hipHostFree(resultHitPanelIds[1]));
													
	CUDA_VALIDATE(hipHostFree(results[0]));
	CUDA_VALIDATE(hipHostFree(results[1]));

}													

/*
Transfer the triangle list and the kd tree nodes to the GPU
*/

void KdTreeGpu::transferNodesToGpu()
{

	gpuTransferTime.start();

	CUDA_VALIDATE(hipMalloc(&this->deviceTrianglesListPtr, trianglesList.size()*sizeof(TrianglePanelPair)));
	CUDA_VALIDATE(hipMemcpyAsync(this->deviceTrianglesListPtr, &this->getTriangle(0), trianglesList.size()*sizeof(TrianglePanelPair), hipMemcpyHostToDevice));

	CUDA_VALIDATE(hipMalloc(&this->deviceNodesPtr, nodesList.size()*sizeof(KdTreeGpuNode)));
	CUDA_VALIDATE(hipMemcpyAsync(this->deviceNodesPtr, &this->getNode(0), nodesList.size()*sizeof(KdTreeGpuNode), hipMemcpyHostToDevice));

	gpuTransferTime.stop();

}

void KdTreeGpu::allocateResultGpuMemory()
{
	CUDA_VALIDATE(hipMalloc(&this->deviceResults, MAX_RAYS_PER_ITERATION*sizeof(GpuResult)));
}

void KdTreeGpu::freeGpuMemory()
{
	if(this->deviceTrianglesListPtr)
	{
		CUDA_VALIDATE(hipFree(this->deviceTrianglesListPtr));
	}

	if(this->deviceNodesPtr)
	{
		CUDA_VALIDATE(hipFree(this->deviceNodesPtr));
	}

	if(this->deviceRayPtr)
	{
		CUDA_VALIDATE(hipFree(this->deviceRayPtr));
	}

	CUDA_VALIDATE(hipFree(deviceHitPanelIdPtr));
	CUDA_VALIDATE(hipFree(deviceResults));
}

void KdTreeGpu::setupCuda()
{
	hipSetDeviceFlags(hipDeviceMapHost);
}